#include "header.cuh"
#define N 16384
int a[N], b[N];

int main() {

    for (int i = 0; i < N; ++i) a[i] = 1;
    int *d_a, *d_b;
    hipMalloc((int **)&d_a, N * 4);
    hipMalloc((int **)&d_b, N * 4);
    hipMemcpy(d_a, a, N * 4, hipMemcpyHostToDevice);
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_a, d_b, N);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_a, d_b, N);
    hipMemcpy(b, d_b, N * 4, hipMemcpyDeviceToHost);
    // for (int i = 0; i < 10; ++i) {
    //     printf("# %d %d\n", N * i / 10, b[N * i / 10]);
    // }
    for (int i = 0; i < N; ++i) {
        if (b[i] != i + 1) {
            printf("# break point is %d\n", i);
            break;
        }
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_temp_storage);
    return 0;
}